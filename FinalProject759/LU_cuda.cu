#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>

using namespace std;
#define TILE 32


// LU Decomposition using Shared Memory 


//Initialize a 2D matrix   
void initialize_matrices(double** a, double** l, double** u, int size){ 
    for (int i = 0; i < size; ++i)
    {
        a[i] = new double[size];
        l[i] = new double[size];
        u[i] = new double[size];
    }
}


// Scale the index for threads to get pivot starting and ending points
__global__ void scaleIndex(double *matrix, int n, int index){
    int start = (index * n + index);
	int end = (index * n + n);
	
	for(int i = start+1; i < end; ++i){
		matrix[i] = (matrix[i] / matrix[start]);
	}

}


// Row elimination Kernel - takes matrix, dimension, currect row index, and block size

__global__ void elim(double *A, int n, int index, int bsize){
	extern __shared__ double pivot[];

	int idThread = threadIdx.x;
	int idBlock = blockIdx.x;
	int blockSize = bsize;

	if(idThread == 0){
	    for(int i = index; i < n; i++){
            pivot[i] = A[(index * n) + i];
        }
	}

	__syncthreads();
    //Variables for pivot, row, start and end
	int pivotRow = (index * n);
	int currentRow = (((blockSize * idBlock) + idThread) * n);
	int start = currentRow + index;
	int end = currentRow + n;
    //If greater than pivot row, loop from start index + 1(next row) to end of column
	if(currentRow > pivotRow){
        for(int i = start+1; i<end; ++i){
            //Set the matrix value of next row and its column - pivot
            A[i] = A[i] - (A[start] * pivot[i-currentRow]);
        }
    }
}





int main(int argc, char** argv){
    int n = atoi(argv[1]);

    //Allocate A matrix, U, and L  for CPU
    double *a = new double[n*n];
    double *ret = new double[n*n];

    srand((unsigned)time(0));
    for(int i=0; i < n * n; i++){
        a[i] = double(-1.0) + (rand()) / ( static_cast <double> (RAND_MAX/2.0));
    }

    
    double *da;
    int numblock = n/TILE + ((n % TILE)?1:0);

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    hipMalloc(&da, n*n* sizeof (double));
    hipMemcpy(da, a, n*n* sizeof(double), hipMemcpyHostToDevice);
    for(int i=0; i<n; ++i){
        scaleIndex<<<1,1>>>(da,n,i);
	    elim<<<numblock,TILE,n*sizeof(double)>>>(da,n,i,TILE);
    }
    hipMemcpy(ret, da, n*n*sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 


    cout << "Runtime for LU Decomposition is: " << ms << endl;

    double** A = new double* [n];
    double** u = new double* [n];
    double** l = new double* [n];
 

    initialize_matrices(A, u, l, n);
    for(int i = 0 ;i < n ; ++i){
        for(int j = 0; j < n; ++j){
            A[i][j] = ret[i*n+j];
        }
    }
    //Take values diagonal values from returned array and pull L and U
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            //Find diagonals
            for(int k=0; k<n; k++){
                //If the outermost for loop is larger or equal to k, then grab L values
                if(i>=k)
                    l[i][k] = A[i][k];
                //Else the rest of the array is zeroes
                else l[i][k] = 0;
                //If loops at diagonal then enter 1 for U, if j > k then we're on upper part of Matrix so fill in values, 
                if(k==j) u[k][j] = 1;
                else if(k<j) u[k][j] = A[k][j];
                else u[k][j] = 0.0;
            }
        }
    }



    hipFree(da);
    hipFree(ret);
    delete[] a;
    delete[] ret; 

    return 0;
}