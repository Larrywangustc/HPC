#include <stdlib.h>             
#include <stdio.h>              
#include <fstream>              
#include <hip/hip_runtime.h>               
#include <iostream>             
#include <iomanip>              
#include <time.h>               
       
using namespace std;            
#define TILE 32                 

__global__ void luDecompositionKernel(float* L, float* U, const float* A, int n){
    // Compute the updated lower and upper triangular matrices
    // at the current thread's index.
    int i = threadIdx.x;
    int j = threadIdx.y;

    for (int k = 0; k < i; ++k){
        U[i * n + j] -= L[i * n + k] * U[k * n + j];
        L[i * n + j] -= L[i * n + k] * L[k * n + j];
    }
    if (i == j){
        L[i * n + i] = 1;
    }
    else{
        L[i * n + j] /= U[j * n + j];
        U[i * n + j] /= U[j * n + j];
    }
}


// Perform LU decomposition using CUDA

void luDecompositionCuda(float* L, float* U, const float* A, int n)
{
    // Allocate device memory for the lower and upper triangular matrices
    float* d_L, *d_U;
    hipMalloc(&d_L, n * n * sizeof(float));
    hipMalloc(&d_U, n * n * sizeof(float));

    // Copy the input matrix to the device
    hipMemcpy(d_L, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_U, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    luDecompositionKernel<<<dim3(n, n), dim3(1, 1), 0>>>(d_L, d_U, A, n);


    // Copy the updated lower and upper triangular matrices from the device
    hipMemcpy(L, d_L, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_L);
    hipFree(d_U);

}
