#include "hip/hip_runtime.h"
#include <stdlib.h>             
#include <stdio.h>              
#include <fstream>              
#include <hip/hip_runtime.h>               
#include <iostream>             
#include <iomanip>              
#include <time.h>               
#include <hip/hip_runtime.h>       
using namespace std;              

#define BSZ 32

__global__ void lu_decomposition_kernel(const float *A, float *L, float *U, const int N)
{
    // Declare thread IDs and block size
    int x = threadIdx.x;
    int y = threadIdx.y;

    extern __shared__ float part_sum[BSZ * BSZ];

    for (int k = 0; k < N; ++k)
    {
        part_sum[y * BSZ + x] = 0;
        for (int i = 0; i < k; ++i)
        {
            part_sum[y * BSZ + x] += A[k * N + i] * A[i * N + y];
        }

        __syncthreads();

        L[k * N + y] = (y == k) ? 1 : A[k * N + y] - part_sum[y * BSZ + x];
        U[k * N + y] = (y == k) ? A[k * N + k] - part_sum[y * BSZ + x] : 0;
    }
}



void luDecompositionCuda(float* L, float* U, const float* A, int n, int block_dim){
    int k = (n - 1) / block_dim + 1;
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(k, k);
    lu_decomposition_kernel<<<dimGrid, dimBlock, block_dim * block_dim * sizeof(float)>>>(A, L, U, n);

}


int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *L, *U;

    A = (float*)malloc(sizeof(float) * n * n);
    L = (float*)malloc(sizeof(float) * n * n);
    U = (float*)malloc(sizeof(float) * n * n);

    float* d_L, *d_U, *d_A;
    hipMalloc(&d_L, n * n * sizeof(float));
    hipMalloc(&d_U, n * n * sizeof(float));
    hipMalloc(&d_A, n * n * sizeof(float));

    srand((unsigned)time(0));
    
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    luDecompositionCuda(d_L, d_U, d_A, n, threads_per_block);
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    
    hipMemcpy(L, d_L, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_L);
    hipFree(d_U);
    hipFree(d_A);

    std::cout << L[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    free(A);
    free(L);
    free(U);
    return 0;
}