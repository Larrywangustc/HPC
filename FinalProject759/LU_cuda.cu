#include "hip/hip_runtime.h"
#include <stdlib.h>             
#include <stdio.h>              
#include <fstream>              
#include <hip/hip_runtime.h>               
#include <iostream>             
#include <iomanip>              
#include <time.h>               
#include <hip/hip_runtime.h>       
using namespace std;              
#define Tile_Width 32
#define BSZ 32

__global__ void lu_decomposition_kernel(const float *A, float *L, float *U, const int N){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int y = by * blockDim.y + ty;
    int x = bx * blockDim.x + tx;
    __shared__ float part_sum[BSZ * BSZ];

    for (int k = 0; k < N; ++k)
    {
        part_sum[y * BSZ + x] = 0;
        for (int i = 0; i < k; ++i)
        {
            part_sum[y * BSZ + x] += A[k * N + i] * A[i * N + y];
        }

        __syncthreads();

        L[k * N + y] = (y == k) ? 1 : A[k * N + y] - part_sum[y * BSZ + x];
        U[k * N + y] = (y == k) ? A[k * N + k] - part_sum[y * BSZ + x] : 0;
    }
}


__global__ void luDecompositionOptimized_kernel(const float* d_A, float* d_L, float* d_U, int N)
{
    // Declare shared memory for the tile of the input matrix
    __shared__ float tile[Tile_Width][Tile_Width];

    // Load the tile of the input matrix into shared memory
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    if (row < N && col < N) {
        tile[ty][tx] = d_A[row * N + col];
    }

    for (int k = 0; k < N; k++) {
        if (row == k) {
            d_L[row * N + col] = 1.0f; 
        }
        if (row > k) {
            d_L[row * N + col] = tile[row][k] / tile[k][k];
            for (int i = k+1; i < Tile_Width; i++) {
                tile[row][i] -= d_L[row * N + col] * tile[k][i];
            }
        }

        if (row <= k) {
            d_U[row * N + col] = tile[row][col];
        }
    }
}


__global__ void lu_decomposition_kernel(const float *A, float *L, float *U, const int N){
    int ty = threadIdx.y;
    int by = blockIdx.y;
    int y = by * blockDim.y + ty;

    if (y >= N) return;  // Check that y is within bounds

    for (int k = 0; k < N; ++k)
    {
        // Calculate the elements of the lower triangular matrix
        for (int i = 0; i < k; ++i)
        {
            L[k * N + y] -= A[k * N + i] * L[i * N + y];
        }

        if (k == y)
        {
            L[k * N + y] = 1;  // Diagonal elements are 1
        }
        else
        {
            L[k * N + y] = A[k * N + y];
        }

        // Calculate the elements of the upper triangular matrix
        for (int i = 0; i < k; ++i)
        {
            U[k * N + y] -= A[k * N + i] * U[i * N + y];
        }

        if (k == y)
        {
            U[k * N + y] = A[k * N + y] / L[k * N + y];  // Diagonal elements are the original value divided by the corresponding element in the lower triangular matrix
        }
        else
        {
            U[k * N + y] = A[k * N + y] / L[k * N + y];
        }
    }
}


void luDecompositionCuda(float* L, float* U, const float* A, int n, int block_dim){
    int k = (n - 1) / block_dim + 1;
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(k, k);
    lu_decomposition_kernel<<<dimGrid, dimBlock>>>(A, L, U, n);
    //luDecompositionOptimized_kernel<<<dimGrid, dimBlock, Tile_Width * Tile_Width * sizeof(float)>>>(A, L, U, n);

}


int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *L, *U;

    A = (float*)malloc(sizeof(float) * n * n);
    L = (float*)malloc(sizeof(float) * n * n);
    U = (float*)malloc(sizeof(float) * n * n);

    float* d_L, *d_U, *d_A;
    hipMalloc(&d_L, n * n * sizeof(float));
    hipMalloc(&d_U, n * n * sizeof(float));
    hipMalloc(&d_A, n * n * sizeof(float));

    srand((unsigned)time(0));
    
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    luDecompositionCuda(d_L, d_U, d_A, n, threads_per_block);
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    
    hipMemcpy(L, d_L, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_L);
    hipFree(d_U);
    hipFree(d_A);

    std::cout << L[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    free(A);
    free(L);
    free(U);
    return 0;
}