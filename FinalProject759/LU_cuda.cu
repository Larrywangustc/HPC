#include <stdlib.h>             
#include <stdio.h>              
#include <fstream>              
#include <hip/hip_runtime.h>               
#include <iostream>             
#include <iomanip>              
#include <time.h>               
       
using namespace std;              
#define Tile_Width 32
#define BSZ 32

__global__ void lu_decomposition_kernel(const float *A, float *L, float *U, const int N)
{
    // Calculate the indices for the thread and block
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int y = by * blockDim.y + ty;
    int x = bx * blockDim.x + tx;
    __shared__ float part_sum[BSZ][BSZ];
    for (int k = 0; k < N; ++k)
    {
        part_sum[ty][tx] = 0;
        for (int i = 0; i < k; ++i)
        {
            part_sum[ty][tx] += A[k * N + i] * A[i * N + y];
        }

        __syncthreads();

        L[k * N + y] = (y == k) ? 1 : A[k * N + y] - part_sum[ty][tx];
        U[k * N + y] = (y == k) ? A[k * N + k] - part_sum[ty][tx] : 0;
    }
}

__global__ void luDecompositionOptimized_kernel(const float* d_A, float* d_L, float* d_U, int N)
{
    // Declare shared memory for the tile of the input matrix
    __shared__ float tile[Tile_Width][Tile_Width];

    // Load the tile of the input matrix into shared memory
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = bx * Tile_Width + tx;
    int col = by * Tile_Width + ty;
    if (row < N && col < N) {
        tile[ty][tx] = d_A[row * N + col];
    }

    // Compute the L and U matrices
    int idx = row * Tile_Width + tx;
    if (idx < N) {
        if (row == idx) {
            d_L[row * N + col] = 1.0f; 
        }
        if (row > idx) {
            d_L[row * N + col] = tile[row][idx] / tile[idx][idx];
            for (int i = idx+1; i < Tile_Width; i++) {
                tile[row][i] -= d_L[row * N + col] * tile[idx][i];
            }
        }

        if (row <= idx) {
            d_U[row * N + col] = tile[row][col];
        }
    }
}


void luDecompositionCuda(float* L, float* U, const float* A, int n, int block_dim){
    int k = (n - 1) / block_dim + 1;
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(k, k);
    lu_decomposition_kernel<<<dimGrid, dimBlock>>>(A, L, U, n);
    //luDecompositionOptimized_kernel<<<dimGrid, dimBlock>>>(A, L, U, n);

}


int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *L, *U;

    A = (float*)malloc(sizeof(float) * n * n);
    L = (float*)malloc(sizeof(float) * n * n);
    U = (float*)malloc(sizeof(float) * n * n);

    float* d_L, *d_U, *d_A;
    hipMalloc(&d_L, n * n * sizeof(float));
    hipMalloc(&d_U, n * n * sizeof(float));
    hipMalloc(&d_A, n * n * sizeof(float));

    srand((unsigned)time(0));
    
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    luDecompositionCuda(d_L, d_U, d_A, n, threads_per_block);
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    
    hipMemcpy(L, d_L, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_L);
    hipFree(d_U);
    hipFree(d_A);

    std::cout << L[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    free(A);
    free(L);
    free(U);
    return 0;
}