#include <iostream>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>                                      
#include <time.h>               
#include <hip/hip_runtime.h>
using namespace std;              

int main(int argc, char **argv){
    const int n = atoi(argv[1]);

    float *A = new float[n * n];
    float *U = new float[n * n];
    float *L = new float[n * n];

    srand((unsigned)time(0));
    
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
        L[i] = 0;
        U[i] = 0;
    }

    thrust::device_vector<float> d_A(n * n);
    //float *d_U;
    //float *d_L;
    thrust::copy(A, A + n * n, d_A.begin());
    thrust::device_vector<float> d_L(n * n);
    thrust::device_vector<float> d_U(n * n);
    thrust::copy(L, L + n * n, d_L.begin());
    thrust::copy(U, U + n * n, d_U.begin());
    //hipMalloc(&d_L, n * n * sizeof(float));
    //hipMalloc(&d_U, n * n * sizeof(float));

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(d_A.begin(), d_A.begin() + n * n)),
        thrust::make_zip_iterator(thrust::make_tuple(d_A.end(), d_A.end() + n * n)),
        d_A.begin(), 
        d_L.begin(), d_U.begin(),
        [n, &d_L, &d_U, &d_A](thrust::tuple<float, float> t) {
            int i = thrust::get<0>(t);
            int j = thrust::get<1>(t);

            if (i == j) {
                return d_L[i * n + j] * d_U[i * n + j];
            } 
            else if (i < j) {
                float sum = 0;
                for (int k = 0; k < i; k++) {
                    sum += d_L[i * n + k] * d_U[k * n + j];
                }
                return (d_A[i * n + j] - sum) / d_U[i * n + i];
            }
            else {
                float sum = 0;
                for (int k = 0; k < j; k++) {
                    sum += d_L[i * n + k] * d_U[k * n + j];
                }
                    return d_A[i * n + j] - sum;
            }
        }
    );

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    std::cout << ms << std::endl;
    std::cout << std::endl;
    return 0;
}