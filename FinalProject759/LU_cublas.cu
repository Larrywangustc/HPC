#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(int argc, char **argv){
    const int n = atoi(argv[1]);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *A;
    hipMalloc((void**)&A, n*n*sizeof(float));
    int *ipiv;
    hipMalloc((void**)&ipiv, n*sizeof(int));
    int info;

    srand((unsigned)time(0));
    
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Perform the LU decomposition
    hipblasSgetrfBatched(handle, n, A, n, ipiv, info, 1);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    std::cout << A[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;

    hipFree(A);
    hipFree(ipiv);
    hipblasDestroy(handle);

    return 0;
}