#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <hipblas.h>

int main(int argc, char **argv){

    const int n = atoi(argv[1]);
    int batchSize = atoi(argv[2]);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipError_t error;
    hipblasStatus_t status;
    float *A;
    error = hipMallocManaged(&A, batchSize * n * n * sizeof(float));
    if (error != hipSuccess) fprintf(stderr,"\nError: %s\n",hipGetErrorString(error));
    srand((unsigned)time(0));

    for(int i=0; i < batchSize * n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    float **Aarray;
    error = hipMallocManaged(&Aarray, batchSize * sizeof(float));
    int *ipiv;
    error = hipMallocManaged(&ipiv, batchSize * n * sizeof(int));
    if (error != hipSuccess) fprintf(stderr,"\nError: %s\n",hipGetErrorString(error));
    int *info;

    error = hipMallocManaged(&info, batchSize * sizeof(int));
    if (error != hipSuccess) fprintf(stderr,"\nError: %s\n",hipGetErrorString(error));
    for (int i = 0; i < batchSize; i++) {
        info[i] = 0;
    }

    for (int i = 0; i < batchSize; i++) {
      Aarray[i] = A + i * n * n;
    }

    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Perform the LU decomposition
    status = hipblasSgetrfBatched(handle, n, (float **)Aarray, n, (int *)ipiv, info, batchSize);
    if (status != HIPBLAS_STATUS_SUCCESS) fprintf(stderr,"error in dgetrf %i\n",status);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    std::cout << A[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;

    hipFree(A);
    hipFree(Aarray);
    hipFree(ipiv);
    hipFree(info);
    hipblasDestroy(handle);

    return 0;
}