#include <stdlib.h>             
#include <stdio.h>              
#include <fstream>              
#include <hip/hip_runtime.h>               
#include <iostream>             
#include <iomanip>                       
       
#include <assert.h>
using namespace std;              
#define Tile_Width 32

__global__ void jacobiUnOptimizedOnDevice(float* x_next_u, float* A_u, float* x_now_u, float* b_u, int N){
    // Optimization step 1: tiling
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        float sigma = 0.0;
        
        int idx_Ai = idx*N;
        
        for (int j=0; j<N; j++)
            if (idx != j)
                sigma += A_u[idx_Ai + j] * x_now_u[j];
        x_next_u[idx] = (b_u[idx] - sigma) / A_u[idx_Ai + idx];
    }
}


__constant__ float b_s[512];
__global__ void jacobiOptimizedOnDevice(float* d_x_next, float* d_A, float* d_x_now,  int N){
    __shared__ float xdsn[Tile_Width];
    __shared__ float xdsx[Tile_Width];

    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int xIndex = bx * Tile_Width + tx;
    
    int idx = xIndex * Tile_Width + threadIdx.x;
    if (idx < N) {
        float sigma = 0.0;
        int idx_Ai = idx * N;
        for (int j=0; j<Tile_Width; j++) {
            if (idx != j) {
                xdsn[tx] = d_x_now[idx*Tile_Width];
                xdsx[tx] = d_x_next[idx * Tile_Width];
                sigma += d_A[idx_Ai + j] * xdsn[tx];           
                xdsx[tx] = (b_s[idx] - sigma) / d_A[idx_Ai + idx];
            }
        }
        for (int k=0; k<N; k++) {
            d_x_next[k* N] = xdsx[tx];
        }
    }
}

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);

    float *A, *b_h, *x_d, *x_next, *x_now;
    float *x_next_u, *x_now_u, *A_u, *b_u;
    float *d_x_now, *d_x_next, *d_A, *b;
    
    int iter = 1000, tileSize = 32;
    
    x_next = (float *) malloc(n * sizeof(float));
    A = (float *) malloc(n * n * sizeof(float));
    x_now = (float *) malloc(n * sizeof(float));
    b_h = (float *) malloc(n * sizeof(float));
    x_d = (float *) malloc(n * sizeof(float));

    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }
    for(int i=0; i < n; i++){
        b_h[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
        x_now[i] = 0;
        x_next[i] = 0;
    }

    assert(hipSuccess == hipMalloc((void **) &x_next_u, n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &A_u, n * n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &x_now_u, n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &b_u, n * sizeof(float)));

    assert(hipSuccess == hipMalloc((void **) &d_x_next, n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &d_A, n * n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &d_x_now, n * sizeof(float)));
    assert(hipSuccess == hipMalloc((void **) &b, n * sizeof(float)));
    
    hipMemcpy(x_next_u, x_next, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(A_u, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x_now_u, x_now, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_u, b_h, n * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_x_next, x_next, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x_now, x_now, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, b_h, n * sizeof(float), hipMemcpyHostToDevice);
    
    int nTiles = n/tileSize + (n%tileSize == 0?0:1);
    int grid = n/tileSize + (n%tileSize == 0?0:1);

    dim3 dGrid(grid, grid),
    dBlock(tileSize, tileSize);
    
    dim3 dimGrid(64,16);
    dim3 dimBlock(16,1);

    hipEvent_t start;
    hipEvent_t stop;
    float ms1;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int k=0; k<iter; k++){
        if (k%2)
            jacobiUnOptimizedOnDevice <<< nTiles, tileSize >>> (x_now_u, A_u, x_next_u, b_u, n);
        else     
            jacobiUnOptimizedOnDevice <<< nTiles, tileSize >>> (x_now_u, A_u, x_next_u, b_u, n);
        hipMemcpy(x_now_u, x_next_u, sizeof(float)*n, hipMemcpyDeviceToDevice);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop); 

    float ms2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int k=0; k<iter; k++){
        if (k%2)
            jacobiOptimizedOnDevice <<< dimGrid, dimBlock >>> (d_x_now,  d_A, d_x_next, n);
        else
            jacobiOptimizedOnDevice <<< dimGrid, dimBlock >>> (d_x_now,  d_A, d_x_next, n);
        hipMemcpy(d_x_now, d_x_next, sizeof(float)*n, hipMemcpyDeviceToDevice);
    }    
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms2, start, stop); 
    
    hipMemcpy(x_d, d_x_next, sizeof(float) * n, hipMemcpyDeviceToHost);

    free(x_next); free(A); free(x_now); free(b_h);
    hipFree(d_x_next); hipFree(d_A); hipFree(d_x_now); hipFree(b);
    hipFree(x_now_u); hipFree(x_next_u); hipFree(A_u); hipFree(b_u);
    

    std::cout << ms1 << std::endl;
    std::cout << ms2 << std::endl;
    std::cout << std::endl;
    return 0;
}
