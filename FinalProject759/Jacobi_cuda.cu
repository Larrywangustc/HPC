#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>


void jacobiOnHost(float* x_next, float* A, float* x_now, float* b_h, int Ni, int Nj)
{
    int i,j;
    float sigma;
    
    for (i=0; i<Ni; i++)
    {
        sigma = 0.0;
        for (j=0; j<Nj; j++)
        {
            if (i != j)
                sigma += A[i*Nj + j] * x_now[j]; // From the
            // argothum sigma is the Rx, and also matrix A is
            // seperated into the Nj + j and Ni + i
        }
        x_next[i] = (b_h[i] - sigma) / A[i*Ni + i];
    }
}

__constant__ float b[512];
__global__ void jacobiConstantOnDevice(float* d_x_next, float* d_A, float* d_x_now,  int Ni, int Nj)
{
    // Optimization step 1: tiling
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < Ni)
    {
        float sigma = 0.0;
        
        int idx_Aci = i*Nj;
        
        for (int j=0; j<Nj; j++)
            if (i != j)
                sigma += d_A[idx_Aci + j] * d_x_now[j];
        d_x_next[i] = (b[i] - sigma) / d_A[idx_Aci + i];
        
    }
}

#define Tile_Width 32
__constant__ float b_s[512];
__global__ void jacobiOptimizedOnDevice(float* d_x_next, float* d_A, float* d_x_now,  int Ni, int Nj)
{
    __shared__ float xdsn[Tile_Width];
    __shared__ float xdsx[Tile_Width];
    // Optimization step 1: tiling
    //read the matrix tile into shared memory
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int xIndex = bx * Tile_Width + tx;
    
    int idx = xIndex * Tile_Width + threadIdx.x;
    if (idx < Ni) {
        float sigma = 0.0;
        int idx_Ai = idx * Nj;
        for (int j=0; j<Tile_Width; j++) {
            if (idx != j) {
                xdsn[tx] = d_x_now[idx*Tile_Width];
                xdsx[tx] = d_x_next[idx * Tile_Width];

                sigma += d_A[idx_Ai + j] * xdsn[tx];
                    
                xdsx[tx] = (b_s[idx] - sigma) / d_A[idx_Ai + idx];
                
  
            }
            
        }
        for (int k=0; k<Ni; k++) {
            d_x_next[k* Ni] = xdsx[tx];
        }   
    }
}