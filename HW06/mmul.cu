#include<hip/hip_runtime.h>
#include<iostream>

#include <hipblas.h>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    float a = 1, b = 0;
    hipblasSgemm(
        handle,
        HIPBLAS_OP_N,//
        HIPBLAS_OP_N,//
        n,
        n,
        n,
        &a,
        A,
        n,
        B,
        n,
        &b,
        C,
        n
        );
    hipDeviceSynchronize();
}