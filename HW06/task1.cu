#include<hip/hip_runtime.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "mmul.h"


int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int n_tests = atoi(argv[2]);
    float *A, *B, *C;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMallocManaged(&A, n * n *sizeof(float));
    hipMallocManaged(&B, n * n *sizeof(float));
    hipMallocManaged(&C, n * n *sizeof(float));
    srand((unsigned)time(0));
    for(int i=0; i < n * n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
        B[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    float ms = 0;
    float sum = 0;
    hipEvent_t start, stop;
    for(int i=0; i<n_tests; i++){
        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
   
        mmul(handle, A, B, C, n);
   
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop); 
        sum += ms;
    }
    sum = sum / n_tests;
    std::cout << sum << std::endl;
    std::cout << std::endl;
    // printf("hipMalloc function : %s\n",hipGetErrorString(hipGetLastError()));
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}
