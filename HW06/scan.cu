#include "hip/hip_runtime.h"
#include "scan.cuh"

__global__ void hillis_steele(const float *g_idata, float *g_odata, int n) {
    extern __shared__ float temp[];
    int thid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pout = 0, pin = 1;
    //temp[thid] = (thid == 0) ? 0: g_idata[thid-1];
    if (i < n){
        temp[thid] = g_idata[i];
    }
    else{
        temp[thid] = 0.0f;
    }
    __syncthreads();
    for(int offset = 1; offset < blockDim.x; offset *= 2 ) {
        pout = 1 - pout; 
        pin  = 1 - pout;
        if(thid >= offset){
            temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid] + temp[pin * blockDim.x + thid - offset];
        }
        else{
            temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid];
        }    
        __syncthreads();
    }
    __syncthreads();
    if(thid == blockDim.x - 1){
        g_odata[blockIdx.x] = temp[thid];
    }
}

__host__ void scan(const float* input, float* output, unsigned int N, unsigned int threads_per_block){
    int n = N;
    int k = (n - 1) / threads_per_block + 1; 
    hillis_steele<<<k, threads_per_block, 2 * sizeof(float) * threads_per_block>>>(input, output, n);
    while(n > 1){
        n = k;
        k = (n - 1) / threads_per_block + 1;
        hillis_steele<<<k, threads_per_block, 2 * sizeof(float) * threads_per_block>>>(output, output, n);
    }
}