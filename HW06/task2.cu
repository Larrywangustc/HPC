#include<hip/hip_runtime.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include "scan.cuh"


int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *B;

    hipMallocManaged(&A, n * sizeof(float));
    hipMallocManaged(&B, n * sizeof(float));
    srand((unsigned)time(0));
    for(int i=0; i < n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    scan(A, B, n, threads_per_block);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    std::cout << B[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    //printf("hipMalloc function : %s\n",hipGetErrorString(hipGetLastError()));
    hipFree(A);
    hipFree(B);
    return 0;
}
