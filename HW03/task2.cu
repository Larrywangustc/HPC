#include<hip/hip_runtime.h>
#include<iostream>

__global__ void simpleKernel(int* data, int a){
    //this adds a value to a variable stored in global memory
    data[threadIdx.x] = a * threadIdx.x + blockIdx.x;
}

int main(){
    const int numElems= 16;
    int hA[numElems], *dA;
    hipMalloc((void**)&dA, sizeof(int) * numElems);
    hipMemset(dA, 0, numElems* sizeof(int));
    
    
    srand((unsigned)time(0));
    int a = int(-10) + (rand()) / ( static_cast <int> (RAND_MAX/20));

    simpleKernel<<<2,8>>>(dA, a);
    //bring the result back from the GPU into the hostArray
    hipMemcpy(&hA, dA, sizeof(int) * numElems, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElems; i++){
        std::cout<< hA[i] << " ";
    }
    //release the memory allocated on the GPU 
    hipFree(dA);
    return 0;
}


