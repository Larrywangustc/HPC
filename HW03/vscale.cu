#include "hip/hip_runtime.h"
#include "vscale.cuh"

__global__ void vscale(const float *a, float *b, unsigned int n){
    if(blockIdx.x * 512 + threadIdx.x < n){
        b[blockIdx.x * 512 + threadIdx.x ] = a[blockIdx.x * 512 + threadIdx.x ] * b[blockIdx.x * 512 + threadIdx.x ];
    }
    
}