#include<hip/hip_runtime.h>
#include<iostream>
#include "vscale.cuh"

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    float hA[n], hB[n];
    //float *hB = new float[n];
    float *dA, *dB;
    hipMalloc((void**)&dA, sizeof(float)* n);
    hipMalloc((void**)&dB, sizeof(float)* n);
    hipMemset(dA, 0, n * sizeof(float));
    hipMemset(dB, 0, n * sizeof(float));

    srand((unsigned)time(0));
    for(int i=0;i<n;i++){
        hA[i] = float(-10) + (rand()) / ( static_cast <float> (RAND_MAX/20));
        hB[i] = (rand()) / ( static_cast <float> (RAND_MAX));
    }
c
    int k = (n - 1) / 512;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
   
    vscale<<<k + 1,512>>>(dA, dB, n); 
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(&hB, dB, sizeof(float) * n, hipMemcpyDeviceToHost);

    std::cout << ms << std::endl;
    std::cout << hB[0] << std::endl;
    std::cout << hB[n - 1] << std::endl;
    std::cout << std::endl;
    hipFree(dA);
    hipFree(dB);
    return 0;
}


