#include<hip/hip_runtime.h>
#include<iostream>
#include "vscale.cuh"

int main(int argc, char **argv){
    unsigned int n = atoi(argv[1]);
    float *a = new float[n];
    float *b = new float[n];

    srand((unsigned)time(0));
    for(int i=0;i<n;i++){
        a[i] = float(-10) + (rand()) / ( static_cast <float> (RAND_MAX/20));
        b[i] = (rand()) / ( static_cast <float> (RAND_MAX));
    }

    int k = (n - 1) / 512;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
   
    vscale<<<k + 1,512>>>(a, b, n); 
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << ms << std::endl;
    std::cout << b[0] << std::endl;
    std::cout << b[n - 1] << std::endl;
    std::cout << std::endl;
    //release the memory allocated on the GPU 
    return 0;
}


