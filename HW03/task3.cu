#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <ratio>
#include "vscale.cuh"

using std::chrono::high_resolution_clock;
using std::chrono::duration;

int main(int argc, char **argv){
    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_sec;

    unsigned int n = atoi(argv[1]);
    float *a = new float[n];
    float *b = new float[n];

    srand((unsigned)time(0));
    for(int i;i<n;i++){
        a[i] = float(-10) + (rand()) / ( static_cast <float> (RAND_MAX/20));
        b[i] = (rand()) / ( static_cast <float> (RAND_MAX));
    }

    int k = (n - 1) / 512;

    start = high_resolution_clock::now();
    vscale<<<k + 1,512>>>(a, b, n);
    end = high_resolution_clock::now();
    
    std::cout << duration_sec.count() << std::endl;
    std::cout << b[0] << std::endl;
    std::cout << b[n - 1] << std::endl;

    //release the memory allocated on the GPU 
    return 0;
}


