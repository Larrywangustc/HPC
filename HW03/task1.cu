#include<hip/hip_runtime.h>
#include<iostream>

__global__ void simpleKernel(int* data){
    int n = 1;
    for(int i=1;i<=threadIdx.x+1;i++){
        n *= i;
    }
    printf("%d!=%d\n", threadIdx.x+1, n);
}

int main(){
    const int numElems= 8;
    int Array[numElems];

    simpleKernel<<<1,numElems>>>(Array);
    hipDeviceSynchronize();
}
