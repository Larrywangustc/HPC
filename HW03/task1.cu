#include<hip/hip_runtime.h>
#include<iostream>

__global__ void simpleKernel(int* data){
    int n = 1;
    for(int i=1;i<=threadIdx.x;i++){
        n *= i;
    }
    std::printf("%d!=%d", threadIdx.x, n);
}

int main(){
    const int numElems= 8;
    int Array[numElems];

    simpleKernel<<<1,numElems>>>(Array);
    hipDeviceSynchronize();
}