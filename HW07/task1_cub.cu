#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    // Set up host arrays
    float h_in[n];
    srand((unsigned)time(0));
    for(int i=0; i < n; i++){
        h_in[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    // Set up device arrays
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
    // Setup device output array
    float* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1));
    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    float gpu_sum;
    CubDebugExit(hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost));

    std::cout << gpu_sum << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    
    return 0;
}