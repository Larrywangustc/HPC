#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/transform.h>
#include<thrust/reduce.h>
#include<thrust/functional.h>
#include<iostream>

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    thrust::host_vector<float> h_vec(n);
    srand((unsigned)time(0));
    for(int i=0; i < n; i++){
        h_vec[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
    }

    thrust::device_vector<float> d_vec(n);
    d_vec = h_vec;
    
    
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float result = thrust::reduce(d_vec.begin(), d_vec.end(), (float) 0, thrust::plus<float>());
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    std::cout << result << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;

}