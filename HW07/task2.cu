#include "count.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/distance.h>
#include <thrust/functional.h>
#include <iostream>

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    thrust::host_vector<int> h_vec(n);
    thrust::device_vector<int> counts(n);
    thrust::device_vector<int> values(n);
    srand((unsigned)time(0));
    for(int i=0; i < n; i++){
        h_vec[i] = (rand() % (501));
    }

    thrust::device_vector<int> d_vec(n);
    d_vec = h_vec;
    
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    count(d_vec, values, counts);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 
    size_t N = values.size();
    std::cout << values[N - 1] << std::endl;
    std::cout << counts[N - 1] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;



}