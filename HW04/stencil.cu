#include "hip/hip_runtime.h"
#include "stencil.cuh"

__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block){
    int k = (n - 1) / threads_per_block + 1;
    int l = threads_per_block * 2 + 4 * R + 1; 
    stencil_kernel<<<k, threads_per_block, l>>>(image, mask, output, n, R);
}

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){
    extern__shared__ int s[];
    float *Image = s;
    float *Mask = (float*)&Image[threads_per_block + 2 * R];
    float *Output = (float*)&Mask[2 * R + 1];  
    int bx = blockIdx.x;
    int tx= threadIdx.x; 
    

    float c = 0;


    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int c = wB* BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB* ty + tx] = Csub;

}