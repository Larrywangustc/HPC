#include "hip/hip_runtime.h"
#include "stencil.cuh"

__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block){
    int k = (n - 1) / threads_per_block + 1;
    int l = threads_per_block + 4 * R + 1; 
    stencil_kernel<<<k, threads_per_block, l*sizeof(float)>>>(image, mask, output, n, R);
}

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){
    extern __shared__ float s[];
    float *Image = s;
    float *Mask = (float*)&Image[blockDim.x + 2 * R];

    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int bd = blockDim.x;
    int N = n;
    int r = R;
    
    Image[tx + r] = image[bx * bd + tx];
    if(tx < r){
        if(bx * bd + tx - r >= 0){
            Image[tx] = image[bx * bd + tx - r];
        }
        else{
            Image[tx] = 1;
        }
        if(bx * bd + tx + bd < N){
            Image[tx + r + bd] = image[bx * bd + tx + bd];
        }
        else{
            Image[tx + r + bd] = 1;
        }
    }
    if((0 <= tx) & (tx < 2 * r + 1)){
        Mask[tx] = mask[tx];
    }
    __syncthreads();
    float c = 0;
    for(int j=-r;j<r+1;j++){
        c += Image[tx + r + j] * Mask[j + r];
    }
    /*for(int j=-r;j<r+1;j++){
        c += image[bx * bd + tx + j] * mask[j + r];
    }*/
    output[bx * bd + tx] = c;
}