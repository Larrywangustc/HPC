#include "hip/hip_runtime.h"
#include "matmul.cuh"
void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){
    int k = (n * n - 1) / threads_per_block + 1;
    matmul_kernel<<<k, threads_per_block>>>(A, B, C, n);
}

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n){
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if(tx < n * n){
        float Cvalue= 0;
        int i = tx / n;
        int j = tx % n;
        for(int k = 0; k < n; ++k)  { 
            Cvalue += A[i * n + k] * B[k * n + j];
        }
        C[i * n + j] = Cvalue;
    }
}