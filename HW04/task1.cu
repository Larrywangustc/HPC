#include<hip/hip_runtime.h>
#include<iostream>
#include "matmul.cuh"

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *B, *C;

    A = (float*)malloc(sizeof(float)* n * n);
    B = (float*)malloc(sizeof(float)* n * n);
    C = (float*)malloc(sizeof(float)* n * n);
    srand((unsigned)time(0));
    for(int i=0; i < n * n; i++){
        A[i] = float(-1) + (rand()) / ( static_cast <float> (RAND_MAX/2));
        B[i] = float(-1) + (rand()) / ( static_cast <float> (RAND_MAX/2));
    }

    float *dA, *dB, *dC;
    hipMalloc((void**)&dA, sizeof(float)* n * n);
    hipMalloc((void**)&dB, sizeof(float)* n * n);
    hipMalloc((void**)&dC, sizeof(float)* n * n);
    hipMemset(dC, 0, n * n * sizeof(float));
    hipMemcpy(dA, A, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    matmul(dA, dB, dC, n, threads_per_block); 
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    hipMemcpy(C, dC, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    
    std::cout << C[n * n - 1] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(A);
    free(B);
    free(C);
    return 0;
}





