#include<hip/hip_runtime.h>
#include<iostream>
#include "stencil.cuh"

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int R = atoi(argv[2]);
    const int threads_per_block = atoi(argv[3]);
    float *image, *mask, *output;

    image = (float*)malloc(sizeof(float)* n);
    mask = (float*)malloc(sizeof(float)* (2 * R + 1));
    output = (float*)malloc(sizeof(float)* n);
    srand((unsigned)time(0));
    for(int i = 0; i < n; i++){
        image[i] = float(-1) + (rand()) / ( static_cast <float> (RAND_MAX/2));
    }
    for(int i = 0; i < (2 * R + 1); i++){
        mask[i] = float(-1) + (rand()) / ( static_cast <float> (RAND_MAX/2));
    }

    float *dA, *dB, *dC;
    hipMalloc((void**)&dA, sizeof(float)* n);
    hipMalloc((void**)&dB, sizeof(float)* (2 * R + 1));
    hipMalloc((void**)&dC, sizeof(float)* n);
    hipMemset(dC, 0, sizeof(float) * n);
    hipMemcpy(dA, image, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(dB, mask, sizeof(float) * (2 * R + 1), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    stencil(dA, dB, dC, n, R, threads_per_block);
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    hipMemcpy(output, dC, sizeof(float) * n, hipMemcpyDeviceToHost);
    
    std::cout << output[n - 1] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(image);
    free(mask);
    free(output);
    return 0;
}





