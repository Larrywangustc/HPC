#include<hip/hip_runtime.h>
#include<iostream>
#include "reduce.cuh"

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int threads_per_block = atoi(argv[2]);
    float *A, *B;
    int k = (n - 1) / threads_per_block + 1; 
    A = (float*)malloc(sizeof(float) * n);
    B = (float*)malloc(sizeof(float) * k);
    srand((unsigned)time(0));
    //float sum = 0;
    for(int i=0; i < n; i++){
        A[i] = float(-1.0) + (rand()) / ( static_cast <float> (RAND_MAX/2.0));
        //sum += A[i];
    }

    float *dA, *dB;
    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMalloc((void**)&dB, sizeof(float) * k);
    hipMemset(dB, 0, k * sizeof(float));
    hipMemcpy(dA, A, sizeof(float) * n, hipMemcpyHostToDevice);
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    reduce(&dA, &dB, n, threads_per_block); 
   
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop); 

    hipMemcpy(B, dB, sizeof(float) * k, hipMemcpyDeviceToHost);
    
    std::cout << B[0] << std::endl;
    std::cout << ms << std::endl;
    std::cout << std::endl;
    hipFree(dA);
    hipFree(dB);
    free(A);
    free(B);
    return 0;
}