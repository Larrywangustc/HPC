#include "hip/hip_runtime.h"
#include "reduce.cuh"

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    if(i + blockDim.x < n){
        sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    }
    else if(i < n){
        sdata[tid] = g_idata[i];
    }
    else{
        sdata[tid] = 0;
    }
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if(tid < s){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    }
}

__host__ void reduce(float **input, float **output, unsigned int N, unsigned int threads_per_block){
    int n = N;
    int k = (n - 1) / threads_per_block + 1; 
    reduce_kernel<<<k, threads_per_block, sizeof(float)*threads_per_block>>>(*input, *output, n);
    while(n > 1){
        n = k;
        k = (n - 1) / threads_per_block + 1;
        reduce_kernel<<<k, threads_per_block, sizeof(float)*threads_per_block>>>(*output, *output, n);
    }
}
                     