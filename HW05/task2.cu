#include<hip/hip_runtime.h>
#include<iostream>
#include "matmul.cuh"

int main(int argc, char **argv){
    const int n = atoi(argv[1]);
    const int block_dim = atoi(argv[2]);
    float *A_float, *B_float, *C_float;
    int *A_int, *B_int, *C_int;
    double *A_double, *B_double, *C_double;

    A_float = (float*)malloc(sizeof(float) * n * n);
    B_float = (float*)malloc(sizeof(float) * n * n);
    C_float = (float*)malloc(sizeof(float) * n * n);
    A_int = (int*)malloc(sizeof(int) * n * n);
    B_int = (int*)malloc(sizeof(int) * n * n);
    C_int = (int*)malloc(sizeof(int) * n * n);
    A_double = (double*)malloc(sizeof(double) * n * n);
    B_double = (double*)malloc(sizeof(double) * n * n);
    C_double = (double*)malloc(sizeof(double) * n * n);
    srand((unsigned)time(0));
    for(int i = 0; i < n * n; i++){
        A_float[i] = 1;
        B_float[i] = 1;
        A_int[i] = 1;
        B_int[i] = 1;
        A_double[i] = 1;
        B_double[i] = 1;
    }

    float *dA_float, *dB_float, *dC_float;
    int *dA_int, *dB_int, *dC_int;
    double *dA_double, *dB_double, *dC_double;
    hipMalloc((void**)&dA_float, sizeof(float) * n * n);
    hipMalloc((void**)&dB_float, sizeof(float) * n * n);
    hipMalloc((void**)&dC_float, sizeof(float) * n * n);
    hipMalloc((void**)&dA_int, sizeof(int) * n * n);
    hipMalloc((void**)&dB_int, sizeof(int) * n * n);
    hipMalloc((void**)&dC_int, sizeof(int) * n * n);
    hipMalloc((void**)&dA_double, sizeof(double) * n * n);
    hipMalloc((void**)&dB_double, sizeof(double) * n * n);
    hipMalloc((void**)&dC_double, sizeof(double) * n * n);
    
    hipMemcpy(dA_float, A_float, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dB_float, B_float, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemset(dC_float, 0, sizeof(float) * n * n);
    hipMemcpy(dA_int, A_int, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dB_int, B_int, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemset(dC_int, 0, sizeof(int) * n * n);
    hipMemcpy(dA_double, A_double, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dB_double, B_double, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemset(dC_double, 0, sizeof(double) * n * n);

    hipEvent_t start;
    hipEvent_t stop;
    float ms_int;
    float ms_float;
    float ms_double;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    matmul_2(dA_float, dB_float, dC_float, n, block_dim);  
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_float, start, stop); 

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    matmul_1(dA_int, dB_int, dC_int, n, block_dim); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_int, start, stop); 

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
   
    matmul_3(dA_double, dB_double, dC_double, n, block_dim); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_double, start, stop); 
    
    hipMemcpy(C_int, dC_int, sizeof(int) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(C_float, dC_float, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(C_double, dC_double, sizeof(double) * n * n, hipMemcpyDeviceToHost);


    /*std::cout << ms_int << ", ";
    std::cout << ms_float << ", ";
    std::cout << ms_double << ", ";
    std::cout << std::endl;*/

    std::cout << C_int[0] << std::endl;
    std::cout << C_int[n * n - 1] << std::endl;
    std::cout << ms_int << std::endl;
    std::cout << C_float[0] << std::endl;
    std::cout << C_float[n * n - 1] << std::endl;
    std::cout << ms_float << std::endl;
    std::cout << C_double[0] << std::endl;
    std::cout << C_double[n * n - 1] << std::endl;
    std::cout << ms_double << std::endl;
    std::cout << std::endl;

    free(A_float);
    free(B_float);
    free(C_float);
    free(A_int);
    free(B_int);
    free(C_int);
    free(A_double);
    free(B_double);
    free(C_double);
    hipFree(dA_int);
    hipFree(dB_int);
    hipFree(dC_int);
    hipFree(dA_float);
    hipFree(dB_float);
    hipFree(dC_float);
    hipFree(dA_double);
    hipFree(dB_double);
    hipFree(dC_double);
    
    return 0;
}

