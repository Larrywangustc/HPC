#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include "matmul.cuh"

__global__ void Matmul_int(const int * A, const int * B, int * C, int N, unsigned int block_dim){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int aBegin = N * block_dim * by;
    int aEnd = aBegin + N - 1;
    int aStep = block_dim;
    int bBegin = block_dim * bx;
    int bStep = block_dim * N;
    int Csub = 0;
    int ai, bi;
    extern __shared__ int S[];
    int *As = S;
    int *Bs = (int*)&As[block_dim * block_dim];
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        ai = a + N * ty + tx;
        bi = b + N * ty + tx;
        if ((ai / N >= N) || (ai % N < a % N) || (a / N + ty >= N) || (a % N + tx >= N)){
            As[tx + ty * block_dim] = 0;
        }
        else{
            As[tx + ty * block_dim] = A[ai];
        }
        if ((bi / N >= N) || (bi % N < b % N) || (b / N + ty >= N) || (b % N + tx >= N)){
            Bs[tx + ty * block_dim] = 0;
        }
        else{
            Bs[tx + ty * block_dim] = B[bi];
        }
        __syncthreads();
        for (int k = 0; k < block_dim; ++k) {
            Csub += As[ty * block_dim + k] * Bs[k * block_dim + tx];
        }
        __syncthreads();
    }
    int c = N * block_dim * by + block_dim * bx;
    if ((aBegin % N + tx < N) && (aBegin / N + ty < N) && (bBegin % N + tx < N) && (bBegin / N + ty < N) && (c + N * ty + tx < N * N)){
        C[c + N * ty + tx] = Csub;
    }
}

__global__ void Matmul_float(const float * A, const float * B, float * C, int N, unsigned int block_dim){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int aBegin = N * block_dim * by;
    int aEnd = aBegin + N - 1;
    int aStep = block_dim;
    int bBegin = block_dim * bx;
    int bStep = block_dim * N;
    float Csub= 0;
    int ai, bi;
    extern __shared__ float S_float[];
    float *As = S_float;
    float *Bs = (float*)&As[block_dim * block_dim];
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        ai = a + N * ty + tx;
        bi = b + N * ty + tx;
        if ((ai / N >= N) || (ai % N < a % N) || (a / N + ty >= N) || (a % N + tx >= N)){
            As[tx + ty * block_dim] = 0;
        }
        else{
            As[tx + ty * block_dim] = A[ai];
        }
        if ((bi / N >= N) || (bi % N < b % N) || (b / N + ty >= N) || (b % N + tx >= N)){
            Bs[tx + ty * block_dim] = 0;
        }
        else{
            Bs[tx + ty * block_dim] = B[bi];
        }
        __syncthreads();
        for (int k = 0; k < block_dim; ++k) {
            Csub += As[ty * block_dim + k] * Bs[k * block_dim + tx];
        }
        __syncthreads();
    }
    int c = N * block_dim * by + block_dim * bx;
    if ((aBegin % N + tx < N) && (aBegin / N + ty < N) && (bBegin % N + tx < N) && (bBegin / N + ty < N) && (c + N * ty + tx < N * N)){
        C[c + N * ty + tx] = Csub;
    }
}

__global__ void Matmul_double(const double * A, const double * B, double * C, int N, unsigned int block_dim){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int aBegin = N * block_dim * by;
    int aEnd = aBegin + N - 1;
    int aStep = block_dim;
    int bBegin = block_dim * bx;
    int bStep = block_dim * N;
    int ai, bi;
    double Csub= 0;
    extern __shared__ double S_double[];
    double *As = S_double;
    double *Bs = (double*)&As[block_dim * block_dim];
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        ai = a + N * ty + tx;
        bi = b + N * ty + tx;
        if ((ai / N >= N) || (ai % N < a % N) || (a / N + ty >= N) || (a % N + tx >= N)){
            As[tx + ty * block_dim] = 0;
        }
        else{
            As[tx + ty * block_dim] = A[ai];
        }
        if ((bi / N >= N) || (bi % N < b % N) || (b / N + ty >= N) || (b % N + tx >= N)){
            Bs[tx + ty * block_dim] = 0;
        }
        else{
            Bs[tx + ty * block_dim] = B[bi];
        }
        __syncthreads();
        for (int k = 0; k < block_dim; ++k) {
            Csub += As[ty * block_dim + k] * Bs[k * block_dim + tx];
        }
        __syncthreads();
    }
    int c = N * block_dim * by + block_dim * bx;
    if ((aBegin % N + tx < N) && (aBegin / N + ty < N) && (bBegin % N + tx < N) && (bBegin / N + ty < N) && (c + N * ty + tx < N * N)){
        C[c + N * ty + tx] = Csub;
    }
    
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    int k = (n - 1) / block_dim + 1;
    dim3 dimGrid(k, k);
    Matmul_int<<<dimGrid, dimBlock, block_dim * block_dim * 2 * sizeof(int)>>>(A, B, C, n, block_dim);
}


__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    int k = (n - 1) / block_dim + 1;
    dim3 dimGrid(k, k);
    Matmul_float<<<dimGrid, dimBlock, block_dim * block_dim * 2 * sizeof(float)>>>(A, B, C, n, block_dim);
}

__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    int k = (n - 1) / block_dim + 1;
    dim3 dimGrid(k, k);
    Matmul_double<<<dimGrid, dimBlock, block_dim * block_dim * 2 * sizeof(double)>>>(A, B, C, n, block_dim);
}